#include "hip/hip_runtime.h"
// This program executes a typical convolutional layer in regular CNNs.Neuron sparsity(zero ratio) is 50% and Weight sparsity is 70%.
#include <iostream>
#include "CNNConvLayer.h"
using namespace std;

int *inGNeu, *inFilt, *outGNeu, *outGGPU; //address for GPU
//declare variables for GPU
void initGPU()
{   
	//allocate memory space(VRAM) on GPU
	hipMalloc(&inGNeu, sizeof(int)* FMSIZE*FMSIZE*FMDEPTH );
	hipMalloc(&inFilt, sizeof(int) * FILTSIZE*FILTSIZE*FMDEPTH*FILTNUM );
	hipMalloc(&outGNeu, sizeof(int)* FILTNUM * FMSIZE * FMSIZE );
	hipMalloc(&outGGPU, sizeof(int)* FILTNUM * FMSIZE/3 * FMSIZE/3 );
	
	//copy data from DRAM to VRAM
	hipMemcpy(inGNeu, inNeu, sizeof(int)* FMSIZE*FMSIZE*FMDEPTH, hipMemcpyHostToDevice);
	hipMemcpy(inFilt, filt, sizeof(int)* FILTSIZE*FILTSIZE*FMDEPTH*FILTNUM, hipMemcpyHostToDevice);
}

// This is the CPU version, please don't modify it
void convLayerCPU()
{
	// declarations for bunch of indexing parameters
	int fn, sli, fmy, fmx, y, x;
	int ifmy, ifmx, ofmy, ofmx;
	int filtIdx, inNeuIdx, outNeuIdx, outIdx;
	int filtVol  = FMDEPTH  * FILTSIZE * FILTSIZE;
	int fmArea   = FMSIZE   * FMSIZE;
	int filtArea = FILTSIZE * FILTSIZE;
	int outArea  = FMSIZE/3 * FMSIZE/3;
	int sum;
	// Convolution
	for(fn = 0; fn < FILTNUM; fn++){
		for(fmy = 0; fmy < FMSIZE; fmy += STRIDE){
			for(fmx = 0; fmx < FMSIZE; fmx += STRIDE){
				sum = 0;
				for(sli = 0; sli < FMDEPTH; sli++){
					for(y = 0; y < FILTSIZE; y++){
						for(x = 0; x < FILTSIZE; x++){
							ifmy = fmy - FILTSIZE / 2 + y;
							ifmx = fmx - FILTSIZE / 2 + x;
							filtIdx = fn*filtVol + sli*filtArea + y*FILTSIZE + x;
							inNeuIdx = sli*fmArea + ifmy*FMSIZE + ifmx;
							if(ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE)
								sum += filt[filtIdx] * inNeu[inNeuIdx];
						}
					}
				}
				// Activation - ReLU
				outNeuIdx = fn*fmArea + fmy*FMSIZE + fmx;
				if(sum <= 0)
					outNeu[outNeuIdx] = 0;
				else
					outNeu[outNeuIdx] = sum;
			}
		}
	}

	// Max Pooling with Window Size 3x3 and stride 3
	int max, tmpVal;
	for(sli = 0; sli < FILTNUM; sli++){
		for(fmy = 0; fmy < FMSIZE/3 ; fmy += 1){
			for(fmx = 0; fmx < FMSIZE/3 ; fmx += 1){
				outNeuIdx = sli*fmArea + fmy*3*FMSIZE + fmx*3;
				max = outNeu[outNeuIdx];
				for(y = 0; y < 3; y++){
					for(x = 0; x < 3; x++){
						ofmy = fmy*3 + y;
						ofmx = fmx*3 + x;
						outNeuIdx = sli*fmArea + ofmy*FMSIZE + ofmx;
						tmpVal = outNeu[outNeuIdx];	
						if(tmpVal > max)
							max = tmpVal;
					}
				}
				outIdx = sli*outArea + fmy*FMSIZE/3 + fmx;
				outCPU[outIdx] = max;
			}
		}
	}
}

/***	Implement your CUDA Kernel here	***/
__global__
void convLayerGPU(int* inGNeu, int* inFilt, int* outGGPU, int* outGNeu) //nonshared
{
	//threads and blocks
	int bx = blockIdx.x; //FILTNUM 128	
	int tx = threadIdx.x; //FMSIZE 27 x(col)
	int ty = threadIdx.y; //FMSIZE 27 y(row)
	
	int sli;
	int ifmy, ifmx, ofmy, ofmx, x, y;
	int filtIdx, inNeuIdx, outNeuIdx, outIdx;
	int filtVol  = FMDEPTH  * FILTSIZE * FILTSIZE; // 96 * 5 * 5
	int fmArea   = FMSIZE   * FMSIZE; // 27 * 27
	int filtArea = FILTSIZE * FILTSIZE; // 5 * 5
	int outArea  = FMSIZE/3 * FMSIZE/3;
	int sum = 0;
	
	//convolution
	for (sli = 0; sli < FMDEPTH; sli++){
		for(y = 0; y < FILTSIZE; y++){ // FILTSIZE 5 y(row)
			for(x = 0; x < FILTSIZE; x++){ // FILTSIZE 5 x(col)
				ifmy = ty - FILTSIZE / 2 + y; //frame_row - 5/2 + filter_row
				ifmx = tx - FILTSIZE / 2 + x; //frame_col - 5/2 + filter_col
				filtIdx = bx*filtVol + sli*filtArea + y*FILTSIZE + x;
				//in_idx = 第幾層*27*27 + frame(row*dim+col)
				inNeuIdx = sli*fmArea + ifmy*FMSIZE + ifmx;
				//inside frame
				if(ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE)
					sum += inFilt[filtIdx] * inGNeu[inNeuIdx];
			}
		}
	}
	
	__syncthreads();
	
	outNeuIdx = bx*fmArea + ty*FMSIZE + tx;
	//reLu
	if(sum <= 0)
		outGNeu[outNeuIdx] = 0;
	else
		outGNeu[outNeuIdx] = sum;	
	
	__syncthreads();
	// Max Pooling with Window Size 3x3 and stride 3
	int max, tmpVal;
	for(sli = 0; sli < FILTNUM; sli++){
		if (ty < FMSIZE/3){
			if (tx < FMSIZE/3){
				outNeuIdx = sli*fmArea + ty*3*FMSIZE + tx*3;
				max = outGNeu[outNeuIdx];
				for(y = 0; y < 3; y++){
					for(x = 0; x < 3; x++){
						ofmy = ty*3 + y;
						ofmx = tx*3 + x;
						outNeuIdx = sli*fmArea + ofmy*FMSIZE + ofmx;
						tmpVal = outGNeu[outNeuIdx];	
						if(tmpVal > max)
							max = tmpVal;
					}
				}
				outIdx = sli*outArea + ty*FMSIZE/3 + tx;
				outGGPU[outIdx] = max;
			}
		}
	}
}
/***	Implement your CUDA Kernel here	***/

int main()
{
	hipSetDevice(2);
	//variables setting and loading input data
	timespec time_begin, time_end; 
	int convLayerCPUExecTime, convLayerGPUExecTime;
	init();
	
	//Convolution by CPU                                                
	clock_gettime(CLOCK_REALTIME, &time_begin);
	convLayerCPU();
	clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerCPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "CPU time for executing a typical convolutional layer = "  <<  ((float)convLayerCPUExecTime)/1000 << "ms" << endl;

	dim3 numBlocks(FILTNUM); //128
	dim3 threadsPerBlock(FMSIZE,FMSIZE); //27*27
	
	//Convolution by GPU
	clock_gettime(CLOCK_REALTIME, &time_begin);
	/***	Lunch your CUDA Kernel here	***/
	initGPU();
	convLayerGPU<<<numBlocks,threadsPerBlock>>>(inGNeu, inFilt, outGGPU, outGNeu); // Lunch the kernel
	hipDeviceSynchronize(); // Do synchronization before clock_gettime()
	hipMemcpy(outGPU, outGGPU , sizeof(int) * FILTNUM * FMSIZE/3 * FMSIZE/3, hipMemcpyDeviceToHost);
	/***	Lunch your CUDA Kernel here	***/
	clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerGPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "GPU time for executing a typical convolutional layer = "  << ((float)convLayerGPUExecTime)/1000 << "ms" << endl;
	
	//hipMemcpy(outNeu_, outGNeu, sizeof(int) * FILTNUM * FMSIZE * FMSIZE, hipMemcpyDeviceToHost);
	
	hipFree(&inGNeu);
	hipFree(&inFilt);
	hipFree(&outGGPU);
	hipFree(&outGNeu);
	//check the anser from CPU and from GPU
	if(checker()){
		cout << "Congratulations! You pass the check." << endl;
		cout << "Speedup: " << (float)convLayerCPUExecTime / convLayerGPUExecTime << endl;
	}
	else
		cout << "Sorry! Your result is wrong." << endl;

	//release memory space
	ending();
	
	return 0;
}
